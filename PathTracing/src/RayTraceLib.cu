#include "hip/hip_runtime.h"
#pragma once
#include "CudaStructs.h"

// ----------------------------------------------------------------------------------
// Operator overloads
// ----------------------------------------------------------------------------------

__host__ __device__ inline float3 operator-(const float3& a, const float3& b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__host__ __device__ inline float3 operator+(const float3& a, const float3& b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__host__ __device__ inline float3 operator*(float s, const float3& v) {
    return make_float3(s * v.x, s * v.y, s * v.z);
}

__device__ __host__ inline float3 operator/(const float3& a, float b) {
    return make_float3(a.x / b, a.y / b, a.z / b);
}

__host__ __device__ inline float3& operator+=(float3& a, const float3& b) {
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
    return a;
}

__host__ __device__ inline float3 operator*(const float3& a, const float3& b) {
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

__host__ __device__ inline float3& operator*=(float3& a, const float3& b) {
    a.x *= b.x; a.y *= b.y; a.z *= b.z;
    return a;
}

__host__ __device__ inline float3 operator*(const float3& v, float s) {
    return make_float3(v.x * s, v.y * s, v.z * s);
}

__device__ __host__ inline float4& operator/=(float4& v, float scalar) {
    float inv = 1.0f / scalar;
    v.x *= inv;
    v.y *= inv;
    v.z *= inv;
    v.w *= inv;
    return v;
}


// ----------------------------------------------------------------------------------
// Math helpers
// ----------------------------------------------------------------------------------

__host__ __device__ inline float dot3(const float3& a, const float3& b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__host__ __device__ inline float3 cross(const float3& a, const float3& b) {
    return make_float3(
        a.y * b.z - a.z * b.y,
        a.z * b.x - a.x * b.z,
        a.x * b.y - a.y * b.x
    );
}

__device__ inline float clamp(float val, float minVal, float maxVal)
{
    return fminf(fmaxf(val, minVal), maxVal);
}

__device__ inline float3 clamp(float3 v, float minVal, float maxVal)
{
    return make_float3(
        clamp(v.x, minVal, maxVal),
        clamp(v.y, minVal, maxVal),
        clamp(v.z, minVal, maxVal)
    );
}

__host__ __device__ inline float3 normalize(const float3& v, float epsilon = 1e-8f) {
    float lenSq = dot3(v, v);
    if (lenSq < epsilon)
        return make_float3(0.0f, 0.0f, 0.0f);
    return v * rsqrtf(lenSq);
}

__host__ __device__ inline uint32_t PackRGBA(uint8_t r, uint8_t g, uint8_t b, uint8_t a = 255) {
    return (static_cast<uint32_t>(a) << 24)
         | (static_cast<uint32_t>(b) << 16)
         | (static_cast<uint32_t>(g) << 8)
         | static_cast<uint32_t>(r);
}

__device__ inline float RandomFloat(uint32_t& seed) {
    seed ^= seed << 13;
    seed ^= seed >> 17;
    seed ^= seed << 5;
    return (seed * 16807 % 2147483647) / 2147483647.0f;
}

__device__ inline float3 RandomInUnitSphere(uint32_t& seed) {
    float3 p;
    do {
        p = make_float3(RandomFloat(seed), RandomFloat(seed), RandomFloat(seed));  
        
        p.x = p.x * 2.0f - 1.0f; 
        p.y = p.y * 2.0f - 1.0f;
        p.z = p.z * 2.0f - 1.0f;
    } while (false); 
    return p;
}

__host__ __device__ inline float3 lerp(const float3& a, const float3& b, float t) {
    return a + t * (b - a);
}

__host__ __device__ inline float3 reflect(const float3& I, const float3& N) {
    return I - 2.0f * dot3(N, I) * N;
}

__device__ inline float4 mul(const float4x4& m, const float4& v) {
    return make_float4(
        m[0][0]*v.x + m[0][1]*v.y + m[0][2]*v.z + m[0][3]*v.w,
        m[1][0]*v.x + m[1][1]*v.y + m[1][2]*v.z + m[1][3]*v.w,
        m[2][0]*v.x + m[2][1]*v.y + m[2][2]*v.z + m[2][3]*v.w,
        m[3][0]*v.x + m[3][1]*v.y + m[3][2]*v.z + m[3][3]*v.w
    );
}


__device__ inline void swap(float& a, float& b) noexcept
{
    const float temp = a;
    a = b;
    b = temp;
}